#include "hip/hip_runtime.h"
#include <ATen/cuda/HIPContext.h>
#include <hip/hip_runtime.h>
#include <cutlass/array.h>
#include <cutlass/cutlass.h>
#include <cutlass/numeric_types.h>
#include <stdio.h>
#include <torch/all.h>

#include <cfloat>
#include <type_traits>
template <typename T, int N>
using AlignedArray = cutlass::AlignedArray<T, N>;
using bfloat16_t = cutlass::bfloat16_t;
using float16_t = cutlass::half_t;
using float32_t = float;

// QQ NOTE: to handle the case for at::Half, error: more than one operator ">" matches these operands: built-in operator
// "arithmetic > arithmetic" function "operator>(const __half &, const __half &)"
template <typename T>
__device__ inline bool cmp_gt(const T& a, const T& b) {
  if constexpr (std::is_same<T, at::Half>::value) {
    // at::Half (or float16_t in our native case) causes ambiguity, so we cast to float.
    return static_cast<float>(a) > static_cast<float>(b);
  } else {
    // For types like float, at::BFloat16, or cutlass::half_t / cutlass::bfloat16_t, assume operator> works as expected.
    return a > b;
  }
}

template <typename T>
__device__ inline bool cmp_eq(const T& a, const T& b) {
  if constexpr (std::is_same<T, at::Half>::value) {
    return static_cast<float>(a) == static_cast<float>(b);
  } else {
    return a == b;
  }
}

// Fixed constants common to both dynamic and static template versions:
static constexpr int WARP_SIZE = 32;
static constexpr int WARPS_PER_CTA = 6;
static constexpr int MAX_VPT = 512;  // maximum VPT we support, > params.VPT = num_expert / num_expert_group

// Create an alias for Array using AlignedArray
template <typename T, int N>
using Array = AlignedArray<T, N>;
// QQ: NOTE expression must have a constant value, this has to be > params.VPT
template <typename T>
using AccessType = AlignedArray<T, MAX_VPT>;

template <typename T>
__device__ inline T recalculate_sigmoid(int expert_idx, T* input_ptr) {
  T val = input_ptr[expert_idx];
  return static_cast<T>(1.0f / (1.0f + expf(-float(val))));
}

// Write the fused expert output for shared experts
template <typename T>
__device__ void write_fused_expert(
    int thread_group_idx,
    int64_t topk,
    int64_t thread_row,
    int64_t topk_excluding_share_expert_fusion,
    int num_fused_shared_experts,
    int num_experts,
    float output_sum,
    double routed_scaling_factor,
    int32_t* indices_ptr,
    float* output_ptr) {
  if (thread_group_idx == 0 && num_fused_shared_experts > 0) {
    int64_t last_idx = topk * thread_row + topk_excluding_share_expert_fusion;
    int64_t expert_offset = 0;
    indices_ptr[last_idx] = static_cast<int32_t>(num_experts + expert_offset);
    output_ptr[last_idx] = output_sum / routed_scaling_factor;
    for (int i = 1; i < num_fused_shared_experts; ++i) {
      ++last_idx;
      ++expert_offset;
      indices_ptr[last_idx] = static_cast<int32_t>(num_experts + expert_offset);
      output_ptr[last_idx] = output_sum / routed_scaling_factor;
    }
  }
  __syncthreads();
}

// Normalize the output for the top-k experts
__device__ void
normalize_output(int thread_group_idx, int64_t topk, int64_t thread_row, float output_sum, float* output_ptr) {
  if (thread_group_idx == 0) {
#pragma unroll
    for (int ii = 0; ii < topk; ++ii) {
      int64_t idx = topk * thread_row + ii;
      output_ptr[idx] = output_ptr[idx] / output_sum;
    }
  }
}

// Reduce the top-k argmax values across threads
template <typename T>
__device__ void topk_argmax_reduce(T& max_val, int& expert, int threads_per_row) {
#pragma unroll
  for (int mask = threads_per_row / 2; mask > 0; mask /= 2) {
    T other_max = static_cast<T>(__shfl_xor_sync(0xFFFFFFFF, static_cast<float>(max_val), mask, threads_per_row));
    int other_expert = __shfl_xor_sync(0xFFFFFFFF, expert, mask, threads_per_row);
    if (cmp_gt(other_max, max_val) || (cmp_eq(other_max, max_val) && other_expert < expert)) {
      max_val = other_max;
      expert = other_expert;
    }
  }
}

// Reduce the top-k argmax values across threads, but for excluding groups
template <typename T>
__device__ void exclude_groups_argmax_reduce(T& max_sum, int& expert, int threads_per_row) {
#pragma unroll
  for (int mask = threads_per_row / 2; mask > 0; mask /= 2) {
    T other_max_sum = static_cast<T>(__shfl_xor_sync(0xFFFFFFFF, static_cast<float>(max_sum), mask, threads_per_row));
    int other_expert = __shfl_xor_sync(0xFFFFFFFF, expert, mask, threads_per_row);
    if (cmp_gt(max_sum, other_max_sum) || (cmp_eq(other_max_sum, max_sum) && other_expert > expert)) {
      max_sum = other_max_sum;
      expert = other_expert;
    }
  }
}

template <typename T, typename Params>
__device__ void moe_fused_gate_small_vpt(
    void* input,
    void* bias,
    float* output_ptr,
    int32_t* indices_ptr,
    int64_t num_rows,
    int64_t topk_group,
    int64_t topk,
    int64_t num_fused_shared_experts,
    double routed_scaling_factor,
    Params params) {
  // Small VPT value optimization path - Using the original non-tile implementation
  // ===== Place the original implementation (non-tile version) code here =====

  int tidx = threadIdx.x;
  int thread_group_idx = threadIdx.y;
  int thread_row = blockIdx.x * params.ROWS_PER_CTA + thread_group_idx * params.ROWS_PER_WARP;
  T* thread_row_ptr = reinterpret_cast<T*>(input) + thread_row * params.NUM_EXPERTS;
  T* bias_ptr = reinterpret_cast<T*>(bias) + thread_row * params.NUM_EXPERTS;
  int first_elt_read_by_thread = tidx * params.VPT;
  int topk_excluding_share_expert_fusion = topk - num_fused_shared_experts;

  Array<T, 32> row_chunk;  // Since VPT ≤ 32, 32 is sufficient.
  Array<T, 32> bias_chunk;

  // Read input data
  T* thread_read_ptr = thread_row_ptr + first_elt_read_by_thread;
  AccessType<T> const* vec_thread_read_ptr = reinterpret_cast<AccessType<T> const*>(thread_read_ptr);

  T* bias_thread_read_ptr = bias_ptr + first_elt_read_by_thread;
  AccessType<T> const* vec_bias_thread_read_ptr = reinterpret_cast<AccessType<T> const*>(bias_thread_read_ptr);

// QQ NOTE: doing the follow will be slower than loop assign and more importantly
// have misaligned address issue when params.VPT < 8 and mismatch with MAX_VPT
// AccessType<T>* row_chunk_vec_ptr = reinterpret_cast<AccessType<T>*>(&row_chunk);
// row_chunk_vec_ptr[0] = vec_thread_read_ptr[0];
#pragma unroll
  for (int ii = 0; ii < params.VPT; ++ii) {
    row_chunk[ii] = vec_thread_read_ptr[0][ii];
    bias_chunk[ii] = vec_bias_thread_read_ptr[0][ii];
  }

  __syncthreads();

////////////////////// Sigmoid //////////////////////
#pragma unroll
  for (int ii = 0; ii < params.VPT; ++ii) {
    row_chunk[ii] = static_cast<T>(1.0f / (1.0f + expf(-float(row_chunk[ii]))));
  }
  __syncthreads();

////////////////////// Add Bias //////////////////////
#pragma unroll
  for (int ii = 0; ii < params.VPT; ++ii) {
    bias_chunk[ii] = row_chunk[ii] + bias_chunk[ii];
  }

////////////////////// Exclude Groups //////////////////////
#pragma unroll
  for (int k_idx = 0; k_idx < params.THREADS_PER_ROW - topk_group;
       ++k_idx) {  // QQ NOTE Here params.THREADS_PER_ROW = num_expert_group
    int expert = first_elt_read_by_thread;
    // local argmax
    T max_val = static_cast<T>(-FLT_MAX);
    T max_val_second = static_cast<T>(-FLT_MAX);
#pragma unroll
    for (int ii = 0; ii < params.VPT; ++ii) {
      T val = bias_chunk[ii];

      if (cmp_gt(val, max_val)) {
        max_val_second = max_val;
        max_val = val;
      } else if (cmp_gt(val, max_val_second)) {
        max_val_second = val;
      }
    }

    // QQ NOTE: currently fixed to pick top2 sigmoid weight value in each expert group and sum them as the group
    // weight to select expert groups
    T max_sum = max_val + max_val_second;

    // argmin reduce
    exclude_groups_argmax_reduce(max_sum, expert, params.THREADS_PER_ROW);

    // clear the max value in the thread
    if (k_idx < params.THREADS_PER_ROW - topk_group) {
      int const thread_to_clear_in_group = expert / params.VPT;

      if (thread_group_idx == thread_to_clear_in_group) {
#pragma unroll
        for (int ii = 0; ii < params.VPT; ++ii) {
          bias_chunk[ii] = static_cast<T>(FLT_MAX);
        }
      }
    }
  }

  __syncthreads();

  ////////////////////// Topk //////////////////////
  float output_sum = 0.0f;
  for (int k_idx = 0; k_idx < topk_excluding_share_expert_fusion; ++k_idx) {
    // local argmax
    T max_val = bias_chunk[0];
    int expert = first_elt_read_by_thread;

    if (!cmp_eq(max_val, static_cast<T>(FLT_MAX))) {
#pragma unroll
      for (int ii = 1; ii < params.VPT; ++ii) {
        T val = bias_chunk[ii];
        if (cmp_gt(val, max_val)) {
          max_val = val;
          expert = first_elt_read_by_thread + ii;
        }
      }
    } else {
      max_val = static_cast<T>(-FLT_MAX);
    }

    // argmax reduce
    topk_argmax_reduce(max_val, expert, params.THREADS_PER_ROW);

    int thread_to_clear_in_group = expert / params.VPT;
    int64_t idx = topk * thread_row + k_idx;

    if (thread_group_idx == thread_to_clear_in_group) {
      int expert_to_clear_in_thread = expert % params.VPT;

      // clear the max value in the thread
      bias_chunk[expert_to_clear_in_thread] = static_cast<T>(-FLT_MAX);

      // store output
      output_ptr[idx] = static_cast<float>(row_chunk[expert_to_clear_in_thread]);
      indices_ptr[idx] = static_cast<int32_t>(expert);
    }

    // accumulate sum for all elements
    if (thread_group_idx == 0) {
      output_sum += output_ptr[idx];
    }

    __syncthreads();
  }

  write_fused_expert<T>(
      thread_group_idx,
      topk,
      thread_row,
      topk_excluding_share_expert_fusion,
      num_fused_shared_experts,
      params.NUM_EXPERTS,
      output_sum,
      routed_scaling_factor,
      indices_ptr,
      output_ptr);

  normalize_output(thread_group_idx, topk, thread_row, output_sum, output_ptr);
}

template <typename T, typename Params>
__device__ void moe_fused_gate_large_vpt(
    void* input,
    void* bias,
    float* output_ptr,
    int32_t* indices_ptr,
    int64_t num_rows,
    int64_t topk_group,
    int64_t topk,
    int64_t num_fused_shared_experts,
    double routed_scaling_factor,
    Params params) {
  // Add shared memory array to store processing results
  // Only allocate shared memory for the currently processed tile, not the entire VPT
  int tidx = threadIdx.x;
  int thread_group_idx = threadIdx.y;
  int thread_row = blockIdx.x * params.ROWS_PER_CTA + thread_group_idx * params.ROWS_PER_WARP;
  T* thread_row_ptr = reinterpret_cast<T*>(input) + thread_row * params.NUM_EXPERTS;
  T* bias_ptr = reinterpret_cast<T*>(bias) + thread_row * params.NUM_EXPERTS;
  int first_elt_read_by_thread = tidx * params.VPT;
  int topk_excluding_share_expert_fusion = topk - num_fused_shared_experts;
  // __shared__ T shared_sigmoid[WARP_SIZE * 32]; // 32 * 32 = 1024
  // __shared__ T shared_bias[WARP_SIZE * 32];    // 32 * 32 = 1024
  __shared__ T shared_sigmoid[WARP_SIZE * (32 + 1)];
  __shared__ T shared_bias[WARP_SIZE * (32 + 1)];

  __shared__ int current_tile_idx;

  // Calculate the offset of the current thread in the warp
  int thread_shared_offset = tidx % WARP_SIZE;

  // Create local arrays for the row chunk and bias chunk and then reinterpret the address of row_chunk as a pointer
  // to AccessType.
  T* thread_read_ptr = thread_row_ptr + first_elt_read_by_thread;
  Array<T, 32> row_chunk;
  AccessType<T> const* vec_thread_read_ptr = reinterpret_cast<AccessType<T> const*>(thread_read_ptr);

  T* bias_thread_read_ptr = bias_ptr + first_elt_read_by_thread;
  Array<T, 32> bias_chunk;
  AccessType<T> const* vec_bias_thread_read_ptr = reinterpret_cast<AccessType<T> const*>(bias_thread_read_ptr);

  // QQ NOTE: doing the follow will be slower than loop assign and more importantly
  // have misaligned address issue when params.VPT < 8 and mismatch with MAX_VPT
  // AccessType<T>* row_chunk_vec_ptr = reinterpret_cast<AccessType<T>*>(&row_chunk);
  // row_chunk_vec_ptr[0] = vec_thread_read_ptr[0];
  // Processing logic: Use a loop to process each tile (32 elements)
  // Find the maximum and second largest values in each tile, then merge the results after processing all tiles.
  T global_max_val = static_cast<T>(-FLT_MAX);
  T global_max_val_second = static_cast<T>(-FLT_MAX);
  int global_max_idx = -1;
  int global_max_second_idx = -1;

#pragma unroll
  for (int tile = 0; tile < (params.VPT + 31) / 32; ++tile) {
    // Synchronize threads to ensure all threads are ready for the next tile
    if (tidx == 0 && threadIdx.y == 0) {
      current_tile_idx = tile;
    }
    __syncthreads();

    int tile_offset = tile * 32;
    int tile_size = min(32, params.VPT - tile_offset);
    if (tile_size <= 0) break;

    // Prefetch the data of the next tile before processing the current one
    if (tile + 1 < (params.VPT + 31) / 32) {
      int next_offset = (tile + 1) * 32;
      int prefetch_size = min(32, params.VPT - next_offset);
      if (prefetch_size > 0) {
#pragma unroll
        for (int i = 0; i < prefetch_size; i += 8) {
          if (std::is_same<T, float32_t>::value) {
            // Prefetching using __ldg for float32 type
            volatile float dummy = __ldg(reinterpret_cast<const float*>(&thread_read_ptr[next_offset + i]));
            volatile float dummy2 = __ldg(reinterpret_cast<const float*>(&bias_thread_read_ptr[next_offset + i]));
          } else {
            // For other types, use volatile to ensure prefetching
            volatile T dummy = thread_read_ptr[next_offset + i];
            volatile T dummy2 = bias_thread_read_ptr[next_offset + i];
          }
        }
      }
    }

// Read row_chunk and bias_chunk
#pragma unroll
    for (int ii = 0; ii < tile_size; ++ii) {
      int global_idx = tile_offset + ii;
      row_chunk[ii] = vec_thread_read_ptr[0][global_idx];
      bias_chunk[ii] = vec_bias_thread_read_ptr[0][global_idx];
    }

// Calculate the maximum and second maximum values in the current tile
#pragma unroll
    for (int ii = 0; ii < tile_size; ++ii) {
      int global_idx = tile_offset + ii;

      // Calculate Sigmoid
      T sigmoid_val = static_cast<T>(1.0f / (1.0f + expf(-float(row_chunk[ii]))));
      // Add bias
      T val_with_bias = sigmoid_val + bias_chunk[ii];

      // Store the result in shared memory
      // int shared_idx = (thread_shared_offset + ii * WARP_SIZE) % (WARP_SIZE * 32);
      int shared_idx = thread_shared_offset + ii * (WARP_SIZE + 1);
      shared_sigmoid[shared_idx] = sigmoid_val;
      shared_bias[shared_idx] = val_with_bias;

      if (cmp_gt(val_with_bias, global_max_val)) {
        global_max_val_second = global_max_val;
        global_max_second_idx = global_max_idx;
        global_max_val = val_with_bias;
        global_max_idx = global_idx;
      } else if (cmp_gt(val_with_bias, global_max_val_second)) {
        global_max_val_second = val_with_bias;
        global_max_second_idx = global_idx;
      }
    }
    __syncthreads();
  }

////////////////////// Exclude Groups //////////////////////
#pragma unroll
  for (int k_idx = 0; k_idx < params.THREADS_PER_ROW - topk_group;
       ++k_idx) {  // QQ NOTE Here params.THREADS_PER_ROW = num_expert_group
    int expert = first_elt_read_by_thread;
    // Use the global maximum instead of recalculating it here
    T max_val = global_max_val;
    T max_val_second = global_max_val_second;

    // QQ NOTE: currently fixed to pick top2 sigmoid weight value in each expert group and sum them as the group
    // weight to select expert groups
    T max_sum = max_val + max_val_second;

    // argmin reduce
    exclude_groups_argmax_reduce(max_sum, expert, params.THREADS_PER_ROW);

    int thread_to_clear_in_group = expert / params.VPT;
    if (thread_group_idx == thread_to_clear_in_group) {
      // Need to clear the global maximum
      int expert_mod = expert % params.VPT;
      if (expert_mod == global_max_idx || expert_mod == global_max_second_idx) {
        int tile_idx = expert_mod / 32;
        int local_idx = expert_mod % 32;
        // Clear in shared memory
        if (tile_idx == current_tile_idx) {
          int shared_idx = (thread_shared_offset + local_idx * WARP_SIZE) % (WARP_SIZE * 32);
          shared_bias[shared_idx] = static_cast<T>(FLT_MAX);
        }

        // Reset global maximum values
        global_max_val = static_cast<T>(-FLT_MAX);
        global_max_val_second = static_cast<T>(-FLT_MAX);
        global_max_idx = -1;
        global_max_second_idx = -1;

// Recalculate the maximum and second maximum values in the current tile
#pragma unroll
        for (int i = 0; i < params.VPT; ++i) {
          int tile_idx = i / 32;
          int local_idx = i % 32;

          T val;
          if (tile_idx == current_tile_idx) {
            int shared_idx = (thread_shared_offset + local_idx * WARP_SIZE) % (WARP_SIZE * 32);
            val = shared_bias[shared_idx];
          } else {
            // Recalculate the value for the current tile
            val = recalculate_sigmoid(i, thread_read_ptr) + bias_thread_read_ptr[i];
          }

          if (cmp_gt(val, global_max_val) && !cmp_eq(val, static_cast<T>(FLT_MAX))) {
            global_max_val_second = global_max_val;
            global_max_second_idx = global_max_idx;
            global_max_val = val;
            global_max_idx = i;
          } else if (cmp_gt(val, global_max_val_second) && !cmp_eq(val, static_cast<T>(FLT_MAX))) {
            global_max_val_second = val;
            global_max_second_idx = i;
          }
        }
      }
    }
  }

  __syncthreads();

  ////////////////////// Topk //////////////////////
  float output_sum = 0.0f;
  for (int k_idx = 0; k_idx < topk_excluding_share_expert_fusion; ++k_idx) {
    // Read the bias value of the current block using local variables for recalculation.
    T local_max_val = static_cast<T>(-FLT_MAX);
    int local_max_idx = -1;
    int expert = first_elt_read_by_thread;

// Determine the current tile index
#pragma unroll
    for (int i = 0; i < params.VPT; ++i) {
      int tile_idx = i / 32;
      int local_idx = i % 32;

      // Read the value from shared memory or recalculate it
      T val;
      if (tile_idx == current_tile_idx) {
        // In the current tile, read directly from shared memory.
        val = shared_bias[thread_shared_offset + local_idx];
      } else {
        // Not in the current tile, recalculating
        int global_offset = i;
        val = recalculate_sigmoid(global_offset, thread_read_ptr) + bias_thread_read_ptr[global_offset];
      }

      if (cmp_gt(val, local_max_val) && !cmp_eq(val, static_cast<T>(FLT_MAX)) &&
          !cmp_eq(val, static_cast<T>(-FLT_MAX))) {
        local_max_val = val;
        local_max_idx = i;
      }
    }

    // If no valid value is found
    if (local_max_idx == -1) {
      local_max_val = static_cast<T>(-FLT_MAX);
    } else {
      // Update the expert index based on the local maximum index
      expert = first_elt_read_by_thread + local_max_idx;
    }

    // argmax reduce
    topk_argmax_reduce(local_max_val, expert, params.THREADS_PER_ROW);

    int thread_to_clear_in_group = expert / params.VPT;
    int64_t idx = topk * thread_row + k_idx;

    if (thread_group_idx == thread_to_clear_in_group) {
      int expert_to_clear_in_thread = expert % params.VPT;
      int tile_idx = expert_to_clear_in_thread / 32;
      int local_idx = expert_to_clear_in_thread % 32;

      // If the current thread is responsible for clearing the expert
      if (tile_idx == current_tile_idx) {
        int shared_idx = (thread_shared_offset + local_idx * WARP_SIZE) % (WARP_SIZE * 32);
        shared_bias[shared_idx] = static_cast<T>(-FLT_MAX);
        output_ptr[idx] = static_cast<float>(shared_sigmoid[shared_idx]);
      } else {
        // Recalculate the sigmoid value for the expert
        output_ptr[idx] = static_cast<float>(recalculate_sigmoid(expert_to_clear_in_thread, thread_read_ptr));
      }

      indices_ptr[idx] = static_cast<int32_t>(expert);
    }

    // accumulate sum for all elements
    if (thread_group_idx == 0) {
      output_sum += output_ptr[idx];
    }

    __syncthreads();
  }

  write_fused_expert<T>(
      thread_group_idx,
      topk,
      thread_row,
      topk_excluding_share_expert_fusion,
      num_fused_shared_experts,
      params.NUM_EXPERTS,
      output_sum,
      routed_scaling_factor,
      indices_ptr,
      output_ptr);

  normalize_output(thread_group_idx, topk, thread_row, output_sum, output_ptr);
}

template <typename T, typename Params>
__global__ void moe_fused_gate_kernel_small_vpt(
    void* input,
    void* bias,
    float* output_ptr,
    int32_t* indices_ptr,
    int64_t num_rows,
    int64_t topk_group,
    int64_t topk,
    int64_t num_fused_shared_experts,
    double routed_scaling_factor,
    Params params) {
  moe_fused_gate_small_vpt<T>(
      input,
      bias,
      output_ptr,
      indices_ptr,
      num_rows,
      topk_group,
      topk,
      num_fused_shared_experts,
      routed_scaling_factor,
      params);
}

template <typename T, typename Params>
__global__ void moe_fused_gate_kernel_large_vpt(
    void* input,
    void* bias,
    float* output_ptr,
    int32_t* indices_ptr,
    int64_t num_rows,
    int64_t topk_group,
    int64_t topk,
    int64_t num_fused_shared_experts,
    double routed_scaling_factor,
    Params params) {
  moe_fused_gate_large_vpt<T>(
      input,
      bias,
      output_ptr,
      indices_ptr,
      num_rows,
      topk_group,
      topk,
      num_fused_shared_experts,
      routed_scaling_factor,
      params);
}

//------------------------------------------------------------------------------
// Templated Kernel Version (using compile-time constants)
//------------------------------------------------------------------------------
template <int VPT_, int NUM_EXPERTS_, int THREADS_PER_ROW_, int ROWS_PER_WARP_, int ROWS_PER_CTA_, int WARPS_PER_CTA_>
struct KernelParams {
  static constexpr int VPT = VPT_;
  static constexpr int NUM_EXPERTS = NUM_EXPERTS_;
  static constexpr int THREADS_PER_ROW = THREADS_PER_ROW_;
  static constexpr int ROWS_PER_WARP = ROWS_PER_WARP_;
  static constexpr int ROWS_PER_CTA = ROWS_PER_CTA_;
  static constexpr int WARPS_PER_CTA = WARPS_PER_CTA_;
};

template <
    typename T,
    int VPT,
    int NUM_EXPERTS,
    int THREADS_PER_ROW,
    int ROWS_PER_WARP,
    int ROWS_PER_CTA,
    int WARPS_PER_CTA>
__global__ void moe_fused_gate_kernel(
    void* input,
    void* bias,
    float* output_ptr,
    int32_t* indices_ptr,
    int64_t num_rows,
    int64_t topk_group,
    int64_t topk,
    int64_t num_fused_shared_experts,
    double routed_scaling_factor) {
  KernelParams<VPT, NUM_EXPERTS, THREADS_PER_ROW, ROWS_PER_WARP, ROWS_PER_CTA, WARPS_PER_CTA> params;
  if constexpr (VPT <= 32) {
    moe_fused_gate_small_vpt<T>(
        input,
        bias,
        output_ptr,
        indices_ptr,
        num_rows,
        topk_group,
        topk,
        num_fused_shared_experts,
        routed_scaling_factor,
        params);
  } else {
    moe_fused_gate_large_vpt<T>(
        input,
        bias,
        output_ptr,
        indices_ptr,
        num_rows,
        topk_group,
        topk,
        num_fused_shared_experts,
        routed_scaling_factor,
        params);
  }
}

// Macro to compute compile-time constants and launch the kernel.
#define LAUNCH_MOE_GATE_CONFIG(T, EXPERTS, EXPERT_GROUP)                                                            \
  do {                                                                                                              \
    constexpr int VPT = (EXPERTS) / (EXPERT_GROUP);                                                                 \
    constexpr int ROWS_PER_WARP = ((EXPERT_GROUP) <= WARP_SIZE) ? (WARP_SIZE / (EXPERT_GROUP)) : 1;                 \
    constexpr int ROWS_PER_CTA = WARPS_PER_CTA * ROWS_PER_WARP;                                                     \
    size_t shared_mem_size = (VPT <= 32) ? 0 : (2 * WARP_SIZE * 32 * sizeof(T) + sizeof(int));                      \
    moe_fused_gate_kernel<T, VPT, (EXPERTS), (EXPERT_GROUP), ROWS_PER_WARP, ROWS_PER_CTA, WARPS_PER_CTA>            \
        <<<static_cast<unsigned int>(num_blocks), block_dim, static_cast<unsigned int>(shared_mem_size), stream>>>( \
            input.data_ptr(),                                                                                       \
            bias.data_ptr(),                                                                                        \
            output.data_ptr<float>(),                                                                               \
            indices.data_ptr<int32_t>(),                                                                            \
            num_rows,                                                                                               \
            topk_group,                                                                                             \
            topk,                                                                                                   \
            num_fused_shared_experts,                                                                               \
            routed_scaling_factor);                                                                                 \
    dispatched = true;                                                                                              \
  } while (0)

// Helper macro for dynamic kernel launch (small/large VPT, all types)
#define LAUNCH_DYNAMIC_MOE_KERNEL(KERNEL, TYPE, SHMEM)                                                  \
  KERNEL<TYPE, KernelParamsDynamic>                                                                     \
      <<<static_cast<unsigned int>(num_blocks), block_dim, static_cast<unsigned int>(SHMEM), stream>>>( \
          input.data_ptr<TYPE>(),                                                                       \
          bias.data_ptr<TYPE>(),                                                                        \
          output.data_ptr<float>(),                                                                     \
          indices.data_ptr<int32_t>(),                                                                  \
          num_rows,                                                                                     \
          topk_group,                                                                                   \
          topk,                                                                                         \
          num_fused_shared_experts,                                                                     \
          routed_scaling_factor,                                                                        \
          params)

//------------------------------------------------------------------------------
// Dynamic Kernel Version (parameters computed at runtime)
//------------------------------------------------------------------------------
struct KernelParamsDynamic {
  int VPT;
  int NUM_EXPERTS;
  int THREADS_PER_ROW;
  int ROWS_PER_WARP;
  int ROWS_PER_CTA;
  int WARPS_PER_CTA;
};

template <typename T>
__global__ void moe_fused_gate_kernel_dynamic(
    void* input,
    void* bias,
    float* output_ptr,
    int32_t* indices_ptr,
    int64_t num_rows,
    int64_t num_experts,
    int64_t num_expert_group,
    int64_t topk_group,
    int64_t topk,
    int64_t num_fused_shared_experts,
    double routed_scaling_factor) {
  KernelParamsDynamic params;
  params.NUM_EXPERTS = num_experts;             // e.g, for deepseek v3, this is 256
  params.VPT = num_experts / num_expert_group;  // e.g., for deepseek v3, this is 256 / 8 = 32
  params.THREADS_PER_ROW = num_expert_group;    // fixed as num_expert_group, e.g., for deepseek v3, this is 8
  params.WARPS_PER_CTA = WARPS_PER_CTA;         // fixed as 6
  params.ROWS_PER_WARP = std::max<int64_t>(1, WARP_SIZE / num_expert_group);  // WARP_SIZE is fixed as 32
  params.ROWS_PER_CTA = params.WARPS_PER_CTA * params.ROWS_PER_WARP;

  if (params.VPT <= 32) {
    moe_fused_gate_small_vpt<T>(
        input,
        bias,
        output_ptr,
        indices_ptr,
        num_rows,
        topk_group,
        topk,
        num_fused_shared_experts,
        routed_scaling_factor,
        params);
  } else {
    moe_fused_gate_large_vpt<T>(
        input,
        bias,
        output_ptr,
        indices_ptr,
        num_rows,
        topk_group,
        topk,
        num_fused_shared_experts,
        routed_scaling_factor,
        params);
  }
}

//------------------------------------------------------------------------------
// Host Launcher Function
//------------------------------------------------------------------------------
std::vector<at::Tensor> moe_fused_gate(
    at::Tensor& input,
    at::Tensor& bias,
    int64_t num_expert_group,
    int64_t topk_group,
    int64_t topk,
    int64_t num_fused_shared_experts,
    double routed_scaling_factor) {
  int64_t num_rows = input.size(0);
  int32_t num_experts = static_cast<int32_t>(input.size(1));
  auto options = torch::TensorOptions().dtype(torch::kFloat32).device(torch::kCUDA);
  auto output = torch::empty({num_rows, topk}, options);
  auto indices = torch::empty({num_rows, topk}, options.dtype(torch::kInt32));

  if (num_expert_group == 1 && num_experts > 128) {
    auto gate = (1.0f / (1.0f + (-input).exp())) + bias;
    auto topk_result = gate.topk(topk, 1, true, true);
    output.copy_(std::get<0>(topk_result));
    indices.copy_(std::get<1>(topk_result));
    return {output, indices};
  }
  // Compute grid dimensions based on runtime value for num_expert_group.
  int64_t rows_per_warp = std::max<int64_t>(1, WARP_SIZE / num_expert_group);
  int64_t num_warps = (num_rows + rows_per_warp - 1) / rows_per_warp;
  int64_t num_blocks = (num_warps + WARPS_PER_CTA - 1) / WARPS_PER_CTA;
  const hipStream_t stream = at::cuda::getCurrentCUDAStream();
  dim3 block_dim(WARP_SIZE, WARPS_PER_CTA);

  // Check 1: Ensure that num_experts is a power of 2, or 384 for Kimi K2.
  TORCH_CHECK(
      (num_experts & (num_experts - 1)) == 0 || num_experts == 384,
      "num_experts must be a power of 2 or 384, but got ",
      num_experts);

  // Check 2: Ensure that num_experts is divisible by num_expert_group. (this also means num_expert_group is power of 2)
  TORCH_CHECK(
      num_experts % num_expert_group == 0,
      "num_experts must be divisible by num_expert_group, but got ",
      num_experts,
      " / ",
      num_expert_group);

  int computed_vpt = static_cast<int>(num_experts / num_expert_group);
  // Check 3: Ensure that num_experts/num_expert_group does not exceed MAX_VPT=512. Maximum VPT indicate max value per
  // threads we can process.
  TORCH_CHECK(
      computed_vpt <= MAX_VPT,
      "Per group experts: num_experts / num_expert_group = (",
      computed_vpt,
      ") exceeds the maximum supported (",
      MAX_VPT,
      ")");

  // Dispatch to templated kernel for known compile-time configurations.
  // We currently only support for:
  //   Case 1: 384 experts, with 1 group.
  //   Case 2: 256 experts, with 8 or 16 groups.
  //   Case 3: 128 experts, with 4 or 8 groups.
  //   Case 4: 64 experts, with 1 groups.
  //   Case 5: other cases, require 8 <= num_experts / num_expert_group <= 64.
  bool dispatched = false;
  switch (num_experts) {
    case 384:
      if (num_expert_group == 1)
        // Kimi K2 config: VPT = 384/1 = 384, ROWS_PER_WARP = 32/1 = 32
        if (input.scalar_type() == at::kBFloat16) {
          LAUNCH_MOE_GATE_CONFIG(bfloat16_t, 384, 1);
        } else if (input.scalar_type() == at::kHalf) {
          LAUNCH_MOE_GATE_CONFIG(float16_t, 384, 1);
        } else if (input.scalar_type() == at::kFloat) {
          LAUNCH_MOE_GATE_CONFIG(float32_t, 384, 1);
        }
      break;
    case 256:
      if (num_expert_group == 8) {
        // This is deepseek v3 case. Here VPT = 256/8 = 32, ROWS_PER_WARP = 32/8 = 4, ROWS_PER_CTA = 6 * 4 = 24.
        if (input.scalar_type() == at::kBFloat16) {
          LAUNCH_MOE_GATE_CONFIG(bfloat16_t, 256, 8);
        } else if (input.scalar_type() == at::kHalf) {
          LAUNCH_MOE_GATE_CONFIG(float16_t, 256, 8);
        } else if (input.scalar_type() == at::kFloat) {
          LAUNCH_MOE_GATE_CONFIG(float32_t, 256, 8);
        }
      } else if (num_expert_group == 16) {
        // Here VPT = 256/16 = 16, ROWS_PER_WARP = 32/16 = 2, ROWS_PER_CTA = 6 * 2 = 12.
        if (input.scalar_type() == at::kBFloat16) {
          LAUNCH_MOE_GATE_CONFIG(bfloat16_t, 256, 16);
        } else if (input.scalar_type() == at::kHalf) {
          LAUNCH_MOE_GATE_CONFIG(float16_t, 256, 16);
        } else if (input.scalar_type() == at::kFloat) {
          LAUNCH_MOE_GATE_CONFIG(float32_t, 256, 16);
        }
      }
      break;
    case 128:
      if (num_expert_group == 4) {
        // VPT = 128/4 = 32, ROWS_PER_WARP = 32/4 = 8, ROWS_PER_CTA = 6 * 8 = 48.
        if (input.scalar_type() == at::kBFloat16) {
          LAUNCH_MOE_GATE_CONFIG(bfloat16_t, 128, 4);
        } else if (input.scalar_type() == at::kHalf) {
          LAUNCH_MOE_GATE_CONFIG(float16_t, 128, 4);
        } else if (input.scalar_type() == at::kFloat) {
          LAUNCH_MOE_GATE_CONFIG(float32_t, 128, 4);
        }
      } else if (num_expert_group == 8) {
        // VPT = 128/8 = 16, ROWS_PER_WARP = 32/8 = 4, ROWS_PER_CTA = 6 * 4 = 24.
        if (input.scalar_type() == at::kBFloat16) {
          LAUNCH_MOE_GATE_CONFIG(bfloat16_t, 128, 8);
        } else if (input.scalar_type() == at::kHalf) {
          LAUNCH_MOE_GATE_CONFIG(float16_t, 128, 8);
        } else if (input.scalar_type() == at::kFloat) {
          LAUNCH_MOE_GATE_CONFIG(float32_t, 128, 8);
        }
      }
      break;
    case 64:
      if (num_expert_group == 1)
        // VPT = 64/1 = 64, ROWS_PER_WARP = 32/1 = 32, ROWS_PER_CTA = 6 * 32 = 192.
        if (input.scalar_type() == at::kBFloat16) {
          LAUNCH_MOE_GATE_CONFIG(bfloat16_t, 64, 1);
        } else if (input.scalar_type() == at::kHalf) {
          LAUNCH_MOE_GATE_CONFIG(float16_t, 64, 1);
        } else if (input.scalar_type() == at::kFloat) {
          LAUNCH_MOE_GATE_CONFIG(float32_t, 64, 1);
        }
      break;
    default:
      break;
  }
  if (!dispatched) {
    // Fallback to the dynamic kernel if none of the supported combinations match.
    // Only support num_experts / num_expert_group <= 512 for dynamic kernels.

    KernelParamsDynamic params;
    params.NUM_EXPERTS = num_experts;
    params.VPT = computed_vpt;
    params.THREADS_PER_ROW = static_cast<int>(num_expert_group);
    params.WARPS_PER_CTA = WARPS_PER_CTA;
    params.ROWS_PER_WARP = static_cast<int>(rows_per_warp);
    params.ROWS_PER_CTA = params.WARPS_PER_CTA * params.ROWS_PER_WARP;

    // Select kernel type and shared memory size according to VPT and dtype
    bool is_small_vpt = computed_vpt <= 32;
    size_t shared_mem_size =
        is_small_vpt                             ? 0
        : (input.scalar_type() == at::kBFloat16) ? (2 * WARP_SIZE * 32 * sizeof(bfloat16_t) + sizeof(int))
        : (input.scalar_type() == at::kHalf)     ? (2 * WARP_SIZE * 32 * sizeof(float16_t) + sizeof(int))
                                                 : (2 * WARP_SIZE * 32 * sizeof(float32_t) + sizeof(int));

    // Launch the correct kernel for each dtype
    if (input.scalar_type() == at::kBFloat16) {
      if (is_small_vpt) {
        // Dynamic branch: VPT <= 32, launch small_vpt kernel
        LAUNCH_DYNAMIC_MOE_KERNEL(moe_fused_gate_kernel_small_vpt, bfloat16_t, shared_mem_size);
      } else {
        // Dynamic branch: VPT > 32, launch large_vpt kernel
        LAUNCH_DYNAMIC_MOE_KERNEL(moe_fused_gate_kernel_large_vpt, bfloat16_t, shared_mem_size);
      }
    } else if (input.scalar_type() == at::kHalf) {
      if (is_small_vpt) {
        LAUNCH_DYNAMIC_MOE_KERNEL(moe_fused_gate_kernel_small_vpt, float16_t, shared_mem_size);
      } else {
        LAUNCH_DYNAMIC_MOE_KERNEL(moe_fused_gate_kernel_large_vpt, float16_t, shared_mem_size);
      }
    } else if (input.scalar_type() == at::kFloat) {
      if (is_small_vpt) {
        LAUNCH_DYNAMIC_MOE_KERNEL(moe_fused_gate_kernel_small_vpt, float32_t, shared_mem_size);
      } else {
        LAUNCH_DYNAMIC_MOE_KERNEL(moe_fused_gate_kernel_large_vpt, float32_t, shared_mem_size);
      }
    } else {
      TORCH_CHECK(false, "Unsupported data type for moe_fused_gate");
    }
  }
  return {output, indices};
}
